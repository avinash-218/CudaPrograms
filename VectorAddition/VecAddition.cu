#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>

using namespace::std;

__global__ void VecAdd(float* a, float* b, float *c, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		c[i] = a[i] + b[i];
}

int main()
{
	int N = 12;
	size_t size = N * sizeof(float);

	// allocate host memory arrays
	float* A = (float*)malloc(size);
	float* B = (float*)malloc(size);
	float* C = (float*)malloc(size);

	// allocate device memory arrays
	float* d_A, * d_B, *d_C;
	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);
	
	for (int i = 0;i < N;i++)
	{
		A[i] = i;
		B[i] = 2*i;
	}

	// copy data from host to device for parallel execution
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	// Kernel invocation
	int threadsPerBlock = 256;
	int blocksPerGrid = (N - 1) / threadsPerBlock + 1;
	VecAdd << < blocksPerGrid, threadsPerBlock>> > (d_A, d_B, d_C, N);

	// copy result from device to host
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	for (int i = 0;i < N;i++)
		cout << A[i] <<B[i]<< C[i] << endl;

	return 0;
}