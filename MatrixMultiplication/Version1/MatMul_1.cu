#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

typedef struct {
	int width;
	int height;
	float* ele;
}Matrix;

__global__ void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// MxN * NxO = MxO
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	float val=0;

	for (int i = 0;i < A.width;i++)
		val += A.ele[row * A.width + i] * B.ele[col + i*B.width];

	C.ele[row * C.width + col] = val;
}

int main()
{
	Matrix A, B, C, d_A, d_B, d_C;	//declare host and device data

	//specify dimension of the matrices
	A.width = 3; A.height = 3;	//3x3
	B.width = 3; B.height = 3;	//3x3
	C.width = 3; C.height = 3;	//3x3

	d_A.width = 3; d_A.height = 3;	//3x3
	d_B.width = 3; d_B.height = 3;	//3x3
	d_C.width = 3; d_C.height = 3;	//3x3

	// dynamic allocation of host data of size of the float matrix
	A.ele = (float*)malloc(A.width * A.height * sizeof(float));
	B.ele = (float*)malloc(B.width * B.height * sizeof(float));
	C.ele = (float*)malloc(C.width * C.height * sizeof(float));

	//initialization of host data
	for (int i = 0;i < A.width * A.height;i++)
		A.ele[i] = float(i + 1);

	for (int i = 0;i < B.width * B.height;i++)
		B.ele[i] = float((i + 1) * 2);

	//dynamic allocation of device data of corresponding sizes
	hipMalloc(&d_A.ele, A.width * A.height * sizeof(float));
	hipMalloc(&d_B.ele, B.width * B.height * sizeof(float));
	hipMalloc(&d_C.ele, C.width * C.height * sizeof(float));

	hipMemcpy(d_A.ele, A.ele, A.width * A.height * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B.ele, B.ele, B.width * B.height * sizeof(float), hipMemcpyHostToDevice);

	// MxN * NxO = MxO
	int num_threads = 16;
	dim3 dimBlock(num_threads, num_threads);	//num_threads x num_threads
	dim3 dimGrid((C.width + dimBlock.x - 1) / dimBlock.x, (C.height + dimBlock.y - 1) / dimBlock.y); //calculate grid size

	MatMul<<<dimGrid, dimBlock >>> (d_A, d_B, d_C);

	hipMemcpy(C.ele, d_C.ele, C.width * C.height * sizeof(float), hipMemcpyDeviceToHost);	//copy data from device to host

	for (int r = 0;r < A.height;r++)
	{
		for (int c = 0;c < A.width;c++)
			cout << A.ele[r * A.width + c] << "\t";
		cout << endl;
	}
	cout << endl;

	for (int r = 0;r < B.height;r++)
	{
		for (int c = 0;c < B.width;c++)
			cout << B.ele[r * B.width + c] << "\t";
		cout << endl;
	}
	cout << endl;

	for (int r = 0;r < C.height;r++)
	{
		for (int c = 0;c < C.width;c++)
			cout << C.ele[r*C.width + c] << "\t";
		cout << endl;
	}

	//free device and host memory
	hipFree(d_A.ele);hipFree(d_B.ele);hipFree(d_C.ele);
	free(A.ele);free(B.ele);free(C.ele);

	return 0;
}