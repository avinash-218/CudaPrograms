#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void MemAccess(float* d_data, size_t pitch, int COL, int ROW)
{
	for (int r = 0;r < ROW;r++)
	{
		float* row = (float*)((char*)d_data + r * pitch);	//get the index in which rth row starts (linear memory with padding denoted by pitch)
		for (int c = 0;c < COL;c++)
			float ele = row[c]; //row[c] => row + c th memory location (pointer arithmetic)
	}
}

int main()
{
	int ROW = 64, COL = 64;
	float* d_data;
	size_t pitch;
	int threadsPerBlock = 512, blocksPerGrid = 100;

	hipMallocPitch(&d_data, &pitch, COL * sizeof(float), ROW);

	MemAccess << < blocksPerGrid, threadsPerBlock >> > (d_data, pitch, COL, ROW);
	return 0;
}